
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void ponderacion(float *output, int n, int l) {
    for (int i = 0; i < n; i++) output[i] = output[i]/l;
}

int main(int argc, char*argv[]) {

    // L: image number, M y N: image shape
    int l, m, n;

    // read firts three importand values
    scanf("%d %d %d", &l, &m, &n);

    //
    int size = m*n;

    float* RhInput = (float*)calloc(size,sizeof(float));
    float* RhOutput = (float*)calloc(size,sizeof(float));

    float* GhInput = (float*)calloc(size,sizeof(float));
    float* GhOutput = (float*)calloc(size,sizeof(float));

    float* BhInput = (float*)calloc(size,sizeof(float));
    float* BhOutput = (float*)calloc(size,sizeof(float));

    int pass=1;

    for(int j=0; j < l; j++){ // iteration for the L images COLOR
        if(pass==1){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n float values // COLOR R
            float aux;
            scanf("%f", &aux);
            RhInput[i]= RhInput[i] + aux;
            }
            pass++;
        }else if (pass==2){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n float values // COLOR G
            float aux;
            scanf("%f", &aux);
            GhInput[i]= GhInput[i] + aux;
            }
            pass++;
        }else if (pass==3){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n float values // COLOR B
            float aux;
            scanf("%f", &aux);
            BhInput[i]= BhInput[i] + aux;
            }
            pass=1;
        }
    

    }

    float* RdInput = NULL, *RdOutput = NULL; // COLOR R
    float* GdInput = NULL, *GdOutput = NULL; // COLOR G
    float* BdInput = NULL, *BdOutput = NULL; // COLOR B


    hipMalloc((void**)&RdInput, sizeof(float)*size);
    hipMalloc((void**)&RdOutput, sizeof(float)*size);

    hipMalloc((void**)&GdInput, sizeof(float)*size);
    hipMalloc((void**)&GdOutput, sizeof(float)*size);

    hipMalloc((void**)&BdInput, sizeof(float)*size);
    hipMalloc((void**)&BdOutput, sizeof(float)*size);


    hipMemcpy(RdInput, RhInput, sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(GdInput, GhInput, sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(BdInput, BhInput, sizeof(float)*size, hipMemcpyHostToDevice);

    ponderacion<<<1,l>>>(RdOutput, size, l);
    ponderacion<<<1,l>>>(GdOutput, size, l);
    ponderacion<<<1,l>>>(BdOutput, size, l);

    hipMemcpy(RhOutput, RdOutput, sizeof(float)*size, hipMemcpyDeviceToHost);
    hipMemcpy(GhOutput, GdOutput, sizeof(float)*size, hipMemcpyDeviceToHost);
    hipMemcpy(BhOutput, BdOutput, sizeof(float)*size, hipMemcpyDeviceToHost);


    printf("\nInput Color R = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",RhInput[i]);
    }

    printf("\nOutput Color R = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",RhOutput[i]);
    }

    printf("\nInput Color G = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",GhInput[i]);
    }

    printf("\nOutput Color G = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",GhOutput[i]);
    }

    printf("\nInput Color B = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",BhInput[i]);
    }

    printf("\nOutput Color B = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",BhOutput[i]);
    }

    printf("\n");
    printf("\n");

    return 0;
}