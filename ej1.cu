#include <stdio.h>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>

__global__ void ponderacion(double *input, double *output, int n, int l) {
    for (int i = 0; i < n; i++) output[i] = input[i]/l;
}

/*
 *  Escritura Archivo
 */
void Write(double* R, double* G, double* B, 
           int M, int N, const char *filename) {
    FILE *fp;
    fp = fopen(filename, "w");
    fprintf(fp, "%d %d\n", M, N);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", R[i]);
    fprintf(fp, "%lf\n", R[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", G[i]);
    fprintf(fp, "%lf\n", G[M*N-1]);
    for(int i = 0; i < M*N-1; i++)
        fprintf(fp, "%f ", B[i]);
    fprintf(fp, "%lf\n", B[M*N-1]);
    fclose(fp);
}

int main(int argc, char*argv[]) {

    // L: image number, M y N: image shape
    int l, m, n;

    // read firts three importand values
    scanf("%d %d %d", &l, &m, &n);

    int size = m*n;

    double* RhInput = (double*)calloc(size,sizeof(double));
    double* RhOutput = (double*)calloc(size,sizeof(double));

    double* GhInput = (double*)calloc(size,sizeof(double));
    double* GhOutput = (double*)calloc(size,sizeof(double));

    double* BhInput = (double*)calloc(size,sizeof(double));
    double* BhOutput = (double*)calloc(size,sizeof(double));

    int pass=1;

    for(int j=0; j < l; j++){ // iteration for the L images COLOR
        if(pass==1){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n double values // COLOR R
            double aux;
            scanf("%lf", &aux);
            RhInput[i]= RhInput[i] + aux;
            }
            pass++;
        }else if (pass==2){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n double values // COLOR G
            double aux;
            scanf("%lf", &aux);
            GhInput[i]= GhInput[i] + aux;
            }
            pass++;
        }else if (pass==3){
            for(int i=0; i < m*n; i++){ // iteration for the line with m*n double values // COLOR B
            double aux;
            scanf("%lf", &aux);
            BhInput[i]= BhInput[i] + aux;
            }
            pass=1;
        }
    

    }

    double* RdInput = NULL, *RdOutput = NULL; // COLOR R
    double* GdInput = NULL, *GdOutput = NULL; // COLOR G
    double* BdInput = NULL, *BdOutput = NULL; // COLOR B


    hipMalloc((void**)&RdInput, sizeof(double)*size);
    hipMalloc((void**)&RdOutput, sizeof(double)*size);

    hipMalloc((void**)&GdInput, sizeof(double)*size);
    hipMalloc((void**)&GdOutput, sizeof(double)*size);

    hipMalloc((void**)&BdInput, sizeof(double)*size);
    hipMalloc((void**)&BdOutput, sizeof(double)*size);


    hipMemcpy(RdInput, RhInput, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(GdInput, GhInput, sizeof(double)*size, hipMemcpyHostToDevice);
    hipMemcpy(BdInput, BhInput, sizeof(double)*size, hipMemcpyHostToDevice);

    ponderacion<<<1,size>>>(RdInput, RdOutput, size, l);
    ponderacion<<<1,size>>>(GdInput, GdOutput, size, l);
    ponderacion<<<1,size>>>(BdInput, BdOutput, size, l);

    hipMemcpy(RhOutput, RdOutput, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipMemcpy(GhOutput, GdOutput, sizeof(double)*size, hipMemcpyDeviceToHost);
    hipMemcpy(BhOutput, BdOutput, sizeof(double)*size, hipMemcpyDeviceToHost);


    /*
    printf("\nOutput Color R = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",RhOutput[i]);
    }

    printf("\nOutput Color G = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",GhOutput[i]);
    }

    printf("\nOutput Color B = ");
    for (int i = 0; i < size; i++) {
        printf("%f\t",BhOutput[i]);
    }
    */

    Write(RhOutput, GhOutput, BhOutput, m, n, "imgGPU.txt");

    free(RhInput);
    free(RhOutput);
    free(GhInput);
    free(GhOutput);
    free(BhInput);
    free(BhOutput);

    hipFree(RdInput);
    hipFree(RdOutput);
    hipFree(GdInput);
    hipFree(GdOutput);
    hipFree(BdInput);
    hipFree(BdOutput);

    printf("\n");
    printf("\n");

    return 0;
}